#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#define THREADS_PER_BLOCK 256
#define ELEMENTS_PER_BLOCK (2 * THREADS_PER_BLOCK)

#define DIV_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

#define FULL_MASK32 0xffffffff
#define FULL_MASK64 0xffffffffffffffff

#ifdef  __CUDA_ARCH__
#define __xx_shfl_down(mask, val, offset) __shfl_down_sync(mask, val, offset)
#elif defined(__HIP_PLATFORM_AMD__) // AMD
#define __xx_shfl_down(mask, val, offset) __shfl_down(val, offset)
#else
#error "Unsupported compiler"
#endif

__device__ float warpReduce(float val) {
  if (warpSize == 32) {
    for (int offset = 16; offset > 0; offset /= 2)
      val += __xx_shfl_down(FULL_MASK32, val, offset);
  }
  if (warpSize == 64) {
    for (int offset = 32; offset > 0; offset /= 2)
      val += __xx_shfl_down(FULL_MASK64, val, offset);

  }
  return val;
}

// TODO: variance is computed by every block
//  each block scales and normalizes only a slice
__global__ void muillm_qkl2norm_fp16_kernel(
    const half* __restrict__ Q, // input = size BxK
    const half* __restrict__ K, // input = size BxK
    half* __restrict__ Q_NORM, // output = size BxK
    half* __restrict__ K_NORM, // output = size BxK
    float epsilon,
    unsigned BQ, // batch size for Q
    unsigned N,
    float scale // 1/K
) {
    int warpCounts = THREADS_PER_BLOCK / warpSize;
    int warpId = threadIdx.x / warpSize;
    int laneId = threadIdx.x % warpSize;

    unsigned B = blockIdx.y;

    const half* __restrict__ X;
    half* __restrict__ Y;

    if (B < BQ) {
        // Q
        X = Q;
        Y = Q_NORM;
    } else {
        // K
        B -= BQ;
        X = K;
        Y = K_NORM;
    }

    // shared state to do the reductions
    __shared__ float shared_acc_var;

    if (threadIdx.x == 0) {
        shared_acc_var = epsilon;
    }
    __syncthreads();

    int current_row = B;
    // align X and Y to the current row
    X = &X[current_row * N];
    Y = &Y[current_row * N];

    // compute the variance (all blocks compute it fully)
    float acc_var = 0.f;
    {
      unsigned nStart = blockIdx.x * ELEMENTS_PER_BLOCK + threadIdx.x * 2;
      // first slice
      {
        unsigned n = nStart;
        for (; n + 1 < N; n += ELEMENTS_PER_BLOCK) {
          float2 x = __half22float2(*((const half2*)&X[n]));
          acc_var += x.x * x.x;
          acc_var += x.y * x.y;
        }
        if (n < N) {
          float x = __half2float(X[n]);
          acc_var += x * x;
        }
      }
      // second slice
      {
        unsigned n = threadIdx.x * 2;
        for (; n + 1 < nStart; n += ELEMENTS_PER_BLOCK) {
          float2 x = __half22float2(*((const half2*)&X[n]));
          acc_var += x.x * x.x;
          acc_var += x.y * x.y;
        }
        if (n < nStart) {
          float x = __half2float(X[n]);
          acc_var += x * x;
        }
      }

      // warp reduce
      acc_var = warpReduce(acc_var);
      // reduce accross warps
      if (laneId == 0) {
          atomicAdd(&shared_acc_var, acc_var);
      }
      __syncthreads();
    }

    // reload reduced sum and finalize variance by computing mean
    float rsqrt_var = rsqrtf(shared_acc_var * scale);

    // normalize & output
    {
      // one thread processes 2 elements
      unsigned n = blockIdx.x * ELEMENTS_PER_BLOCK + threadIdx.x * 2;
      if (n + 1 < N) {
        float2 x = __half22float2(*((const half2*)&X[n]));

        float yx = (x.x * rsqrt_var);
        float yy = (x.y * rsqrt_var);
        
        Y[n + 0] = __float2half(yx);
        Y[n + 1] = __float2half(yy);
      }
      if (n < N) {
        float x = __half2float(X[n]);

        float y = (x * rsqrt_var);
        
        Y[n] = __float2half(y);
      }
    }
}

void muillm_qkl2norm_fp16(
  hipStream_t stream,
  unsigned BQ,
  unsigned BK,
  unsigned N,
  const half* q,
  const half* k,
  half* q_norm,
  half* k_norm,
  float epsilon
) {
  const int threads_per_blocks = THREADS_PER_BLOCK;
  // launch enough blocks to cover all elements in q and k
  const dim3 num_blocks = dim3(DIV_ROUND_UP(N, ELEMENTS_PER_BLOCK), BQ+BK, 1);

  float scale = 1.f / N;

  muillm_qkl2norm_fp16_kernel<<<num_blocks, threads_per_blocks, 0, stream>>>(
    q,
    k,
    q_norm,
    k_norm,
    epsilon,
    BQ,
    N,
    scale
  );
}