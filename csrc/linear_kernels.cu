#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include <hip/hip_fp16.h>

#define ROWS_PER_BLOCK 4
#define THREADS_PER_BLOCK 64

#define DIV_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

#define FULL_MASK32 0xffffffff
#define FULL_MASK64 0xffffffffffffffff

#ifdef  __CUDA_ARCH__
#define __xx_shfl_down(mask, val, offset) __shfl_down_sync(mask, val, offset)
#elif defined(__HIP_PLATFORM_AMD__) // AMD
#define __xx_shfl_down(mask, val, offset) __shfl_down(val, offset)
#else
#error "Unsupported compiler"
#endif

__device__ float warpReduce(float val) {
  if (warpSize == 32) {
    for (int offset = 16; offset > 0; offset /= 2)
      val += __xx_shfl_down(FULL_MASK32, val, offset);
  }
  if (warpSize == 64) {
    for (int offset = 32; offset > 0; offset /= 2)
      val += __xx_shfl_down(FULL_MASK64, val, offset);

  }
  return val;
}

static inline void __device__ dot2(float& acc, const float2& a, const float2& b) {
  acc += a.x * b.x;
  acc += a.y * b.y;
}

struct __align__(8) half4 {
    half x;
    half y;
    half z;
    half w;
};

static inline float4 __device__ __half42float4(const half4& v) {
  float4 f;
  f.x = __half2float(v.x);
  f.y = __half2float(v.y);
  f.z = __half2float(v.z);
  f.w = __half2float(v.w);

  return f;
}

static inline void __device__ dot4(float& acc, const float4& a, const float4& b) {
  acc += a.x * b.x;
  acc += a.y * b.y;
  acc += a.z * b.z;
  acc += a.w * b.w;
}

template <typename T>
static inline const T* __device__ addr(const T* p, unsigned index) {
  // helps the AMDGPU compiler understand it can use the sgrp pair + single vgpr addressing mode
  unsigned byte_offset = sizeof(T) * index;
  const uint8_t* p8 = (const uint8_t*)p;
  return (const T*) (p8 + byte_offset);
}

__global__ void muillm_gemv_kernel(
    const half* __restrict__ W, // weight matrix - size N x K
    const half* __restrict__ B, // optional bias - size N
    const half* __restrict__ X, // input = size K
    half* __restrict__ Y, // output - size N
    unsigned N,
    unsigned K
) {
  int warpCounts = THREADS_PER_BLOCK / warpSize;
  int warpId = threadIdx.x / warpSize;
  int laneId = threadIdx.x % warpSize;

  // can process ROWS_PER_BLOCK rows
  // shared state to do the reductions
  __shared__ float shared_accs[ROWS_PER_BLOCK];

  // initialize the shared memory
  if (threadIdx.x < ROWS_PER_BLOCK) {
    shared_accs[threadIdx.x] = 0.f;
  }
  if (THREADS_PER_BLOCK > warpSize) {
    __syncthreads();
  }

  {
    int current_row = blockIdx.x * ROWS_PER_BLOCK + 0;
    if (current_row + 3 < N) {

      // compute the t-th element of Y. by doing the dot product with the
      // t-th row of W
      const half* W0 = &W[(current_row + 0) * K];
      const half* W1 = &W[(current_row + 1) * K];
      const half* W2 = &W[(current_row + 2) * K];
      const half* W3 = &W[(current_row + 3) * K];

      float acc0 = 0.f;
      float acc1 = 0.f;
      float acc2 = 0.f;
      float acc3 = 0.f;

      // do the dot product
      {
        unsigned k; // should be 2 * tidx ?
        //*
        for (k = threadIdx.x * 2; k + 1 < K; k += (THREADS_PER_BLOCK * 2)) {
          // vectorized
          float2 x = __half22float2(*((const half2*)addr(X, k)));
          float2 w0 = __half22float2(*((const half2*)addr(W0, k)));
          float2 w1 = __half22float2(*((const half2*)addr(W1, k)));
          float2 w2 = __half22float2(*((const half2*)addr(W2, k)));
          float2 w3 = __half22float2(*((const half2*)addr(W3, k)));

          dot2(acc0, w0, x);
          dot2(acc1, w1, x);
          dot2(acc2, w2, x);
          dot2(acc3, w3, x);
        }
        if (k < K) {
          // remainder
          float x = __half2float(*addr(X,k));
          float w0 = __half2float(*addr(W0,k));
          float w1 = __half2float(*addr(W1,k));
          float w2 = __half2float(*addr(W2,k));
          float w3 = __half2float(*addr(W3,k));
          acc0 += w0 * x;
          acc1 += w1 * x;
          acc2 += w2 * x;
          acc3 += w3 * x;
        }
      }

      // warp reduce
      acc0 = warpReduce(acc0);
      acc1 = warpReduce(acc1);
      acc2 = warpReduce(acc2);
      acc3 = warpReduce(acc3);

      // reduce accross warps
      if (laneId == 0) {
        atomicAdd(&shared_accs[0], acc0);
        atomicAdd(&shared_accs[1], acc1);
        atomicAdd(&shared_accs[2], acc2);
        atomicAdd(&shared_accs[3], acc3);
      }
    } else {
      for (int i = 0; i < ROWS_PER_BLOCK; i++) {
        // compute the t-th element of Y. by doing the dot product with the
        // t-th row of W
        int current_row = blockIdx.x * ROWS_PER_BLOCK + i;

        if (current_row >= N)
          break;

        const half* W_ = &W[current_row * K];
      
        // do the dot product
        float acc = 0.f;
        for (int k = threadIdx.x; k < K; k += THREADS_PER_BLOCK) {
          float w = __half2float(W_[k]);
          acc += w * __half2float(X[k]);
        }

        // warp reduce
        acc = warpReduce(acc);

        // reduce accross warps
        if (laneId == 0) {
          atomicAdd(&shared_accs[i], acc);
        }
      }
    }
  }

  if (THREADS_PER_BLOCK > warpSize) {
    __syncthreads();
  }

  // write out the results
  {
    if (threadIdx.x >= ROWS_PER_BLOCK)
      return;

    int current_row = blockIdx.x * ROWS_PER_BLOCK + threadIdx.x;

    if (current_row < N) {
      float acc = shared_accs[threadIdx.x]; // read the fully reduced value
      if (B != nullptr) { // add the bias first if there is one
        acc += __half2float(B[current_row]);
      }

      // write the output value
      Y[current_row] = __float2half(acc);
    }
  }
}

at::Tensor muillm_linear_forward_cuda(
    torch::Tensor& weights,
    torch::Tensor* bias,
    torch::Tensor& x) {

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const auto N = weights.size(0);
  const auto K = weights.size(1);

  auto dtype = torch::kFloat16;
  auto output_options = at::TensorOptions()
                            .dtype(dtype)
                            .layout(at::kStrided)
                            .device(at::kCUDA)
                            .requires_grad(false);

  // y has the same dimensions as x, except the last dim that is given by
  // the out_features of weights
  auto output_sizes = x.sizes().vec();
  output_sizes[output_sizes.size() - 1] = N;

  auto y = torch::empty(output_sizes, output_options);

  const int threads_per_blocks = THREADS_PER_BLOCK;
  const int num_blocks = DIV_ROUND_UP(N, ROWS_PER_BLOCK);

  muillm_gemv_kernel<<<num_blocks, threads_per_blocks, 0, stream>>>(
    (const half*)weights.data_ptr(),
    bias == nullptr ? nullptr : (const half*)bias->data_ptr(),
    (const half*)x.data_ptr(),
    (half*)y.data_ptr(),
    N,
    K
  );

  return y;
}

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

at::Tensor muillm_linear_forward(
    torch::Tensor weights,
    torch::Tensor bias,
    torch::Tensor x) {
  //return torch::addmm(bias, x, weights.transpose(0, 1));
  return muillm_linear_forward_cuda(weights, &bias, x);
}

at::Tensor muillm_linear_forward_no_bias(
    torch::Tensor weights,
    torch::Tensor x) {
  CHECK_INPUT(weights);
  CHECK_INPUT(x);

  return muillm_linear_forward_cuda(weights, nullptr, x);
  //return torch::matmul(x, weights.transpose(0, 1));
}