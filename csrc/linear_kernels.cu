#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include <hip/hip_fp16.h>

#include "linear_kernels.cuh"

// Python trampoline

at::Tensor muillm_linear_forward_trampoline(
  muillm_engine_ptr engine,
  torch::Tensor x,
  torch::Tensor weights,
  std::optional<torch::Tensor> norm_weights_,
  float epsilon,
  std::optional<torch::Tensor> mul_bias_,
  std::optional<torch::Tensor> add_bias_,
  std::optional<torch::Tensor> residual_) {
  auto undef_tensor = torch::Tensor();

  torch::Tensor norm_weights = norm_weights_.has_value() ? norm_weights_.value() : undef_tensor;
  torch::Tensor mul_bias = mul_bias_.has_value() ? mul_bias_.value() : undef_tensor;
  torch::Tensor add_bias = add_bias_.has_value() ? add_bias_.value() : undef_tensor;
  torch::Tensor residual = residual_.has_value() ? residual_.value() : undef_tensor;
  return muillm_linear_activ_forward(
      engine.engine_ptr,
      norm_weights,
      epsilon,
      weights,
      mui_activation::Identity,
      mul_bias,
      add_bias,
      residual,
      x
  );
}

//
// actual module
//

#define ROWS_PER_BLOCK 4
#define GEMV_THREADS_PER_BLOCK 64

#define DIV_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

#define FULL_MASK32 0xffffffff
#define FULL_MASK64 0xffffffffffffffff

#ifdef  __CUDA_ARCH__
#define __xx_shfl_down(mask, val, offset) __shfl_down_sync(mask, val, offset)
#elif defined(__HIP_PLATFORM_AMD__) // AMD
#define __xx_shfl_down(mask, val, offset) __shfl_down(val, offset)
#else
#error "Unsupported compiler"
#endif

__device__ float warpReduce(float val) {
  if (warpSize == 32) {
    for (int offset = 16; offset > 0; offset /= 2)
      val += __xx_shfl_down(FULL_MASK32, val, offset);
  }
  if (warpSize == 64) {
    for (int offset = 32; offset > 0; offset /= 2)
      val += __xx_shfl_down(FULL_MASK64, val, offset);

  }
  return val;
}

static inline void __device__ dot2(float& acc, const float2& a, const float2& b) {
  acc += a.x * b.x;
  acc += a.y * b.y;
}

struct __align__(8) half4 {
    half x;
    half y;
    half z;
    half w;
};

static inline float4 __device__ __half42float4(const half4& v) {
  float4 f;
  f.x = __half2float(v.x);
  f.y = __half2float(v.y);
  f.z = __half2float(v.z);
  f.w = __half2float(v.w);

  return f;
}

static inline void __device__ dot4(float& acc, const float4& a, const float4& b) {
  acc += a.x * b.x;
  acc += a.y * b.y;
  acc += a.z * b.z;
  acc += a.w * b.w;
}

template <typename T>
static inline const T* __device__ addr(const T* p, unsigned index) {
  // helps the AMDGPU compiler understand it can use the sgrp pair + single vgpr addressing mode
  unsigned byte_offset = sizeof(T) * index;
  const uint8_t* p8 = (const uint8_t*)p;
  return (const T*) (p8 + byte_offset);
}

static inline float __device__ silu(float x) {
  return x / (1.0f + expf(-x));
}

template<int THREADS_PER_BLOCK>
__global__ void muillm_gemv_kernel(
    const half* __restrict__ W, // weight matrix - size N x K
    const half* __restrict__ X, // input = size K
    mui_activation activation, // activation function 
    const half* __restrict__ MB, // optional multiplicative bias - size N (applied before additive bias)
    const half* __restrict__ AB, // optional additive bias - size N
    const half* __restrict__ RB, // optional residual - size N
    half* __restrict__ Y, // output - size N
    unsigned N,
    unsigned K
) {
  int warpCounts = THREADS_PER_BLOCK / warpSize;
  int warpId = threadIdx.x / warpSize;
  int laneId = threadIdx.x % warpSize;

  // can process ROWS_PER_BLOCK rows
  // shared state to do the reductions
  __shared__ float shared_accs[ROWS_PER_BLOCK];

  // initialize the shared memory
  if (threadIdx.x < ROWS_PER_BLOCK) {
    shared_accs[threadIdx.x] = 0.f;
  }
  if (THREADS_PER_BLOCK > warpSize) {
    __syncthreads();
  }

  {
    int current_row = blockIdx.x * ROWS_PER_BLOCK + 0;
    if (current_row + 3 < N) {

      // compute the t-th element of Y. by doing the dot product with the
      // t-th row of W
      const half* W0 = &W[(current_row + 0) * K];
      const half* W1 = &W[(current_row + 1) * K];
      const half* W2 = &W[(current_row + 2) * K];
      const half* W3 = &W[(current_row + 3) * K];

      float acc0 = 0.f;
      float acc1 = 0.f;
      float acc2 = 0.f;
      float acc3 = 0.f;

      // do the dot product
      {
        unsigned k; // should be 2 * tidx ?
        //*
        for (k = threadIdx.x * 2; k + 1 < K; k += (THREADS_PER_BLOCK * 2)) {
          // vectorized
          float2 x = __half22float2(*((const half2*)addr(X, k)));
          float2 w0 = __half22float2(*((const half2*)addr(W0, k)));
          float2 w1 = __half22float2(*((const half2*)addr(W1, k)));
          float2 w2 = __half22float2(*((const half2*)addr(W2, k)));
          float2 w3 = __half22float2(*((const half2*)addr(W3, k)));

          dot2(acc0, w0, x);
          dot2(acc1, w1, x);
          dot2(acc2, w2, x);
          dot2(acc3, w3, x);
        }
        if (k < K) {
          // remainder
          float x = __half2float(*addr(X,k));
          float w0 = __half2float(*addr(W0,k));
          float w1 = __half2float(*addr(W1,k));
          float w2 = __half2float(*addr(W2,k));
          float w3 = __half2float(*addr(W3,k));
          acc0 += w0 * x;
          acc1 += w1 * x;
          acc2 += w2 * x;
          acc3 += w3 * x;
        }
      }

      // warp reduce
      acc0 = warpReduce(acc0);
      acc1 = warpReduce(acc1);
      acc2 = warpReduce(acc2);
      acc3 = warpReduce(acc3);

      // reduce accross warps
      if (laneId == 0) {
        atomicAdd(&shared_accs[0], acc0);
        atomicAdd(&shared_accs[1], acc1);
        atomicAdd(&shared_accs[2], acc2);
        atomicAdd(&shared_accs[3], acc3);
      }
    } else {
      for (int i = 0; i < ROWS_PER_BLOCK; i++) {
        // compute the t-th element of Y. by doing the dot product with the
        // t-th row of W
        int current_row = blockIdx.x * ROWS_PER_BLOCK + i;

        if (current_row >= N)
          break;

        const half* W_ = &W[current_row * K];
      
        // do the dot product
        float acc = 0.f;
        {
          for (int k = threadIdx.x; k < K; k += THREADS_PER_BLOCK) {
            float w = __half2float(W_[k]);
            float x = __half2float(X[k]);
            acc += w * x;
          }
        }


        // warp reduce
        acc = warpReduce(acc);

        // reduce accross warps
        if (laneId == 0) {
          atomicAdd(&shared_accs[i], acc);
        }
      }
    }
  }

  if (THREADS_PER_BLOCK > warpSize) {
    __syncthreads();
  }

  // write out the results
  {
    if (threadIdx.x >= ROWS_PER_BLOCK)
      return;

    int current_row = blockIdx.x * ROWS_PER_BLOCK + threadIdx.x;

    if (current_row < N) {
      float acc = shared_accs[threadIdx.x]; // read the fully reduced value

      if (activation == mui_activation::Silu) {
        // apply the activation if there is one
        acc = silu(acc);
      }

      if (MB != nullptr) { // apply the multipicative bias if there is one
        acc *= __half2float(MB[current_row]);
      }

      if (AB != nullptr) { // apply the additive bias if there is one
        acc += __half2float(AB[current_row]);
      }
      if (RB != nullptr) { // apply the residual if there is one
        acc += __half2float(RB[current_row]);
      }
      // write the output value
      Y[current_row] = __float2half(acc);
    }
  }
}

template<int THREADS_PER_BLOCK>
__global__ void muillm_gemv_norm_inputs_kernel(
    const half* __restrict__ NW, // input normalization weights matrix - size K
    const half* __restrict__ W, // weight matrix - size N x K
    const half* __restrict__ X, // input = size K
    mui_activation activation, // activation function 
    const half* __restrict__ MB, // optional multiplicative bias - size N (applied before additive bias)
    const half* __restrict__ AB, // optional additive bias - size N
    const half* __restrict__ RB, // optional residual - size N
    half* __restrict__ Y, // output - size N
    unsigned N,
    unsigned K,
    float epsilon,
    float scale
) {
  int warpCounts = THREADS_PER_BLOCK / warpSize;
  int warpId = threadIdx.x / warpSize;
  int laneId = threadIdx.x % warpSize;

  float var_x = 0.f;

  // can process ROWS_PER_BLOCK rows
  // shared state to do the reductions
  __shared__ float shared_accs[ROWS_PER_BLOCK];
  __shared__ float shared_var_x;

  // initialize the shared memory
  if (threadIdx.x < ROWS_PER_BLOCK) {
    shared_accs[threadIdx.x] = 0.f;
  }
  if (threadIdx.x == 0) {
    shared_var_x = epsilon;
  }
  if (THREADS_PER_BLOCK > warpSize) {
    __syncthreads();
  }

  {
    int current_row = blockIdx.x * ROWS_PER_BLOCK + 0;
    if (current_row + 3 < N) {

      // compute the t-th element of Y. by doing the dot product with the
      // t-th row of W
      const half* W0 = &W[(current_row + 0) * K];
      const half* W1 = &W[(current_row + 1) * K];
      const half* W2 = &W[(current_row + 2) * K];
      const half* W3 = &W[(current_row + 3) * K];

      float acc0 = 0.f;
      float acc1 = 0.f;
      float acc2 = 0.f;
      float acc3 = 0.f;

      // do the dot product
      {
        // need to normalize the inputs
  
        unsigned k; // should be 2 * tidx ?
        //*
        for (k = threadIdx.x * 2; k + 1 < K; k += (THREADS_PER_BLOCK * 2)) {
          // vectorized
          float2 x = __half22float2(*((const half2*)addr(X, k)));
          float2 nw = __half22float2(*((const half2*)addr(NW, k)));

          float2 w0 = __half22float2(*((const half2*)addr(W0, k)));
          float2 w1 = __half22float2(*((const half2*)addr(W1, k)));
          float2 w2 = __half22float2(*((const half2*)addr(W2, k)));
          float2 w3 = __half22float2(*((const half2*)addr(W3, k)));

          // accumulate for the variance
          dot2(var_x, x, x);

          // multiply with normalization weights
          x.x = x.x * nw.x;
          x.y = x.y * nw.y;

          dot2(acc0, w0, x);
          dot2(acc1, w1, x);
          dot2(acc2, w2, x);
          dot2(acc3, w3, x);
        }
        if (k < K) {
          // remainder
          float x = __half2float(*addr(X,k));
          float nw = __half2float(*addr(NW,k));


          float w0 = __half2float(*addr(W0,k));
          float w1 = __half2float(*addr(W1,k));
          float w2 = __half2float(*addr(W2,k));
          float w3 = __half2float(*addr(W3,k));

          // accumulate for the variance
          var_x += x * x;

          // multiply with normalization weights
          x *= nw;

          acc0 += w0 * x;
          acc1 += w1 * x;
          acc2 += w2 * x;
          acc3 += w3 * x;
        }
      }

      // warp reduce
      var_x = warpReduce(var_x);
      acc0 = warpReduce(acc0);
      acc1 = warpReduce(acc1);
      acc2 = warpReduce(acc2);
      acc3 = warpReduce(acc3);

      // reduce accross warps
      if (laneId == 0) {
        atomicAdd(&shared_var_x, var_x);
        atomicAdd(&shared_accs[0], acc0);
        atomicAdd(&shared_accs[1], acc1);
        atomicAdd(&shared_accs[2], acc2);
        atomicAdd(&shared_accs[3], acc3);
      }
    } else {
      for (int i = 0; i < ROWS_PER_BLOCK; i++) {
        // compute the t-th element of Y. by doing the dot product with the
        // t-th row of W
        int current_row = blockIdx.x * ROWS_PER_BLOCK + i;

        if (current_row >= N)
          break;

        const half* W_ = &W[current_row * K];
      
        // do the dot product
        float acc = 0.f;
        if (i == 0) {
          // accumulate the variance
          for (int k = threadIdx.x; k < K; k += THREADS_PER_BLOCK) {
            float w = __half2float(W_[k]);

            float x = __half2float(X[k]);
            float nw = __half2float(NW[k]);

            // accumuate the variance
            var_x += x * x;

            // multiply with normalization weights
            x *= nw;

            acc += w * x;
          }
        } else {
          for (int k = threadIdx.x; k < K; k += THREADS_PER_BLOCK) {
            float w = __half2float(W_[k]);

            float x = __half2float(X[k]);
            float nw = __half2float(NW[k]);

            // don't accumulate the variance (we already have done it with i == 0)

            // multiply with normalization weights
            x *= nw;

            acc += w * x;
          }
        }


        // warp reduce
        var_x = warpReduce(var_x);
        acc = warpReduce(acc);

        // reduce accross warps
        if (laneId == 0) {
          atomicAdd(&shared_var_x, var_x);
          atomicAdd(&shared_accs[i], acc);
        }
      }
    }
  }

  if (THREADS_PER_BLOCK > warpSize) {
    __syncthreads();
  }

  // write out the results
  {
    float rsqrt_var = rsqrtf(shared_var_x * scale);

    if (threadIdx.x >= ROWS_PER_BLOCK)
      return;

    int current_row = blockIdx.x * ROWS_PER_BLOCK + threadIdx.x;

    if (current_row < N) {
      float acc = shared_accs[threadIdx.x] * rsqrt_var; // read the fully reduced value and scale

      if (activation == mui_activation::Silu) {
        // apply the activation if there is one
        acc = silu(acc);
      }

      if (MB != nullptr) { // apply the multipicative bias if there is one
        acc *= __half2float(MB[current_row]);
      }

      if (AB != nullptr) { // apply the additive bias if there is one
        acc += __half2float(AB[current_row]);
      }
      if (RB != nullptr) { // apply the residual if there is one
        acc += __half2float(RB[current_row]);
      }
      // write the output value
      Y[current_row] = __float2half(acc);
    }
  }
}

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

void muillm_linear_activ_forward_placed_output(
    muillm_engine_t* engine,
    torch::Tensor& norm_weights,
    float epsilon,
    torch::Tensor& weights,
    mui_activation activ,
    torch::Tensor& mul_bias,
    torch::Tensor& add_bias,
    torch::Tensor& residual,
    torch::Tensor& x,
    void* output_ptr,
    hipStream_t stream) {
  bool normalize = norm_weights.defined();
  if (normalize) {
    CHECK_INPUT(norm_weights);
  }
  CHECK_INPUT(weights);
  if (mul_bias.defined()) {
    CHECK_INPUT(mul_bias);
  }
  if (add_bias.defined()) {
    CHECK_INPUT(add_bias);
  }
  if (residual.defined()) {
    CHECK_INPUT(residual);
  }
  CHECK_INPUT(x);

  const auto N = weights.size(0);
  const auto K = weights.size(1);

  const int num_blocks = DIV_ROUND_UP(N, ROWS_PER_BLOCK);
  int threads_per_blocks = GEMV_THREADS_PER_BLOCK;

  int simd_lanes = engine->gpu_infos[0]->simd_lanes;

  // try to occupy enough to saturate memory bandwidth
  while ((num_blocks * threads_per_blocks < 8 * simd_lanes) && threads_per_blocks < 256) {
    threads_per_blocks *= 2;
  }

  if (normalize) {
    const auto NORM_K = norm_weights.size(0);
    TORCH_CHECK(K == NORM_K, "fused normalization is not supported when sharding on dim 1 (K != NORM_K)");

    float scale = 1.f / K;

    if (threads_per_blocks == 64) {
      muillm_gemv_norm_inputs_kernel<64><<<num_blocks, threads_per_blocks, 0, stream>>>(
        norm_weights.defined() ? (const half*)norm_weights.data_ptr() : nullptr,
        (const half*)weights.data_ptr(),
        (const half*)x.data_ptr(),
        activ,
        mul_bias.defined() ? (const half*)mul_bias.data_ptr() : nullptr,
        add_bias.defined() ? (const half*)add_bias.data_ptr() : nullptr,
        residual.defined() ? (const half*)residual.data_ptr() : nullptr,
        (half*) output_ptr,
        N,
        K,
        epsilon,
        scale
      );
    } else if (threads_per_blocks == 128) {
      muillm_gemv_norm_inputs_kernel<128><<<num_blocks, threads_per_blocks, 0, stream>>>(
        norm_weights.defined() ? (const half*)norm_weights.data_ptr() : nullptr,
        (const half*)weights.data_ptr(),
        (const half*)x.data_ptr(),
        activ,
        mul_bias.defined() ? (const half*)mul_bias.data_ptr() : nullptr,
        add_bias.defined() ? (const half*)add_bias.data_ptr() : nullptr,
        residual.defined() ? (const half*)residual.data_ptr() : nullptr,
        (half*) output_ptr,
        N,
        K,
        epsilon,
        scale
      );
    } else if (threads_per_blocks == 256) {
      muillm_gemv_norm_inputs_kernel<256><<<num_blocks, threads_per_blocks, 0, stream>>>(
        norm_weights.defined() ? (const half*)norm_weights.data_ptr() : nullptr,
        (const half*)weights.data_ptr(),
        (const half*)x.data_ptr(),
        activ,
        mul_bias.defined() ? (const half*)mul_bias.data_ptr() : nullptr,
        add_bias.defined() ? (const half*)add_bias.data_ptr() : nullptr,
        residual.defined() ? (const half*)residual.data_ptr() : nullptr,
        (half*) output_ptr,
        N,
        K,
        epsilon,
        scale
      );
    } else {
      TORCH_CHECK(false, "unsupported threads_per_blocks");
    }
  } else {

    if (threads_per_blocks == 64) {
      muillm_gemv_kernel<64><<<num_blocks, threads_per_blocks, 0, stream>>>(
        (const half*)weights.data_ptr(),
        (const half*)x.data_ptr(),
        activ,
        mul_bias.defined() ? (const half*)mul_bias.data_ptr() : nullptr,
        add_bias.defined() ? (const half*)add_bias.data_ptr() : nullptr,
        residual.defined() ? (const half*)residual.data_ptr() : nullptr,
        (half*) output_ptr,
        N,
        K
      );
    } else if (threads_per_blocks == 128) {
      muillm_gemv_kernel<128><<<num_blocks, threads_per_blocks, 0, stream>>>(
        (const half*)weights.data_ptr(),
        (const half*)x.data_ptr(),
        activ,
        mul_bias.defined() ? (const half*)mul_bias.data_ptr() : nullptr,
        add_bias.defined() ? (const half*)add_bias.data_ptr() : nullptr,
        residual.defined() ? (const half*)residual.data_ptr() : nullptr,
        (half*) output_ptr,
        N,
        K
      );
    } else if (threads_per_blocks == 256) {
      muillm_gemv_kernel<256><<<num_blocks, threads_per_blocks, 0, stream>>>(
        (const half*)weights.data_ptr(),
        (const half*)x.data_ptr(),
        activ,
        mul_bias.defined() ? (const half*)mul_bias.data_ptr() : nullptr,
        add_bias.defined() ? (const half*)add_bias.data_ptr() : nullptr,
        residual.defined() ? (const half*)residual.data_ptr() : nullptr,
        (half*) output_ptr,
        N,
        K
      );
    } else {
      TORCH_CHECK(false, "unsupported threads_per_blocks");
    }
  }
}

at::Tensor muillm_linear_activ_forward(
    muillm_engine_t* engine,
    torch::Tensor& norm_weights,
    float epsilon,
    torch::Tensor& weights,
    mui_activation activ,
    torch::Tensor& mul_bias,
    torch::Tensor& add_bias,
    torch::Tensor& residual,
    torch::Tensor& x) {
  CHECK_INPUT(x);

  auto device = x.device();
  hipStream_t stream = at::cuda::getCurrentCUDAStream(device.index());

  const auto N = weights.size(0);

  auto dtype = torch::kFloat16;
  auto output_options = at::TensorOptions()
                            .dtype(dtype)
                            .layout(at::kStrided)
                            .device(device) // same output device as inputs
                            .requires_grad(false);

  // y has the same dimensions as x, except the last dim that is given by
  // the out_features of weights
  auto output_sizes = x.sizes().vec();
  output_sizes[output_sizes.size() - 1] = N;

  auto y = torch::empty(output_sizes, output_options);

  void* output_ptr = y.data_ptr();

  muillm_linear_activ_forward_placed_output(
    engine,
    norm_weights,
    epsilon,
    weights,
    activ,
    mul_bias,
    add_bias,
    residual,
    x,
    output_ptr,
    stream
  );

  return y;
}